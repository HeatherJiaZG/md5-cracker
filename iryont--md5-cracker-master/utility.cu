#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/device_functions.h>

#include <stdint.h>
#include <iostream>

#include "consts.h"

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true){
  if(code != hipSuccess){
    std::cout << "Error: " << hipGetErrorString(code) << " " << file << " " << line << std::endl;
    if(abort){
      exit(code);
    }
  }
}

__device__ __host__ bool next(uint8_t* length, char* word, uint32_t increment){
  uint32_t idx = 0;
  uint32_t add = 0;
  
  while(increment > 0 && idx < CONST_WORD_LIMIT){
    if(idx >= *length && increment > 0){
      increment--;
    }
    
    add = increment + word[idx];
    word[idx] = add % CONST_CHARSET_LENGTH;
    increment = add / CONST_CHARSET_LENGTH;
    idx++;
  }
  
  if(idx > *length){
    *length = idx;
  }
  
  if(idx > CONST_WORD_LENGTH_MAX){
    return false;
  }

  return true;
}

