#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/device_functions.h>

#include <stdint.h>
#include <iostream>

#include "consts.h"

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true){
  if(code != hipSuccess){
    std::cout << "Error: " << hipGetErrorString(code) << " " << file << " " << line << std::endl;
    if(abort){
      exit(code);
    }
  }
}

void getHashBins(char* target, uint32_t* hashes) {
  for(uint8_t i = 0; i < 4; i++){
    char tmp[16];
    
    strncpy(tmp, target + i * 8, 8);
    sscanf(tmp, "%x", &hashes[i]);  
    uint32_t hash1 = (hashes[i] & 0xFF000000) >> 24;
    uint32_t hash2 = (hashes[i] & 0x00FF0000) >> 8;
    uint32_t hash3 = (hashes[i] & 0x0000FF00) << 8;
    uint32_t hash4 = (hashes[i] & 0x000000FF) << 24;
    hashes[i] = hash1 | hash2 | hash3 | hash4;
  }
}

__device__ __host__ bool next(uint8_t* length, char* word, uint32_t increment){
  uint32_t idx = 0;
  uint32_t add = 0;
  
  while(increment > 0 && idx < CONST_WORD_LIMIT){
    if(idx >= *length && increment > 0){
      increment--;
    }
    
    add = increment + word[idx];
    word[idx] = add % CONST_CHARSET_LENGTH;
    increment = add / CONST_CHARSET_LENGTH;
    idx++;
  }
  
  if(idx > *length){
    *length = idx;
  }
  
  if(idx > CONST_WORD_LENGTH_MAX){
    return false;
  }

  return true;
}

