#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/device_functions.h>

#include <stdint.h>
#include <iostream>

#include "consts.h"

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true){
  if(code != hipSuccess){
    std::cout << "Error: " << hipGetErrorString(code) << " " << file << " " << line << std::endl;
    if(abort){
      exit(code);
    }
  }
}

void getHashBins(char* target, UINT32* hashes) {
  for(int i = 0; i < 4; i++){
    char tmp[16];
    
    strncpy(tmp, target + i * 8, 8);
    sscanf(tmp, "%x", &hashes[i]);  
    UINT32 hash1 = (hashes[i] & 0xFF000000);
    hash1 = hash1 >> 24;
    UINT32 hash2 = (hashes[i] & 0x00FF0000);
    hash2 = hash2 >> 8;
    UINT32 hash3 = (hashes[i] & 0x0000FF00);
    hash3 = hash3 << 8;
    UINT32 hash4 = (hashes[i] & 0x000000FF);
    hash4 = hash4 << 24;
    hashes[i] = hash1 | hash2 | hash3 | hash4;
  }
}

__device__ __host__ bool advance_step(uint8_t* length, char* word, UINT32 increment){
  int i = 0;
  UINT32 add = 0;
  for (i = 0; i<CONST_WORD_LIMIT; i ++) {
    if (increment <= 0) {
      break;
    }
    if(i >= *length && increment > 0){
      increment--;
    }
    add = increment + word[i];
    word[i] = add % CONST_CHARSET_LENGTH;
    increment = add / CONST_CHARSET_LENGTH;
  }

  if(i > *length){
    *length = i;
  }
  if(i > CONST_WORD_LENGTH_MAX){
    return false;
  }
  return true;
}

