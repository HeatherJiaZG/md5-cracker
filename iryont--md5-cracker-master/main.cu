#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <time.h>
#include <string.h>
#include <stdlib.h>
#include <stdint.h>
#include <sstream>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/device_functions.h>

#include "consts.h"
#include "utility.cu"
#include "lib_md5.cu"
#include "lib_md5.h"

char g_word[CONST_WORD_LIMIT];
char g_charset[] = "abcdefg";
char g_cracked[CONST_WORD_LIMIT];

__device__ char g_deviceCharset[CONST_CHARSET_LENGTH];
__device__ char g_deviceCracked[CONST_WORD_LIMIT];

struct deviceInfo{
	struct hipDeviceProp_t prop; // Device Properties
	int id; // Device ID
	int max_threads; // Device max threads per block
	int max_blocks; // Device max blocks
	int global_memory_len;
};


#define REQUIRED_SHARED_MEMORY 64
#define FUNCTION_PARAM_ALLOC 256
struct deviceInfo device;


__global__ void md5Crack(uint8_t wordLength, char* charsetWord, UINT32 hash01, UINT32 hash02, UINT32 hash03, UINT32 hash04){
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  
  /* Shared variables */
  __shared__ char sharedCharset[CONST_CHARSET_LENGTH];
  for(int i = 0; i < CONST_CHARSET_LENGTH; i++){
    sharedCharset[i] = g_deviceCharset[i];
  }
  
  /* Thread variables */
  uint8_t threadWordLength = wordLength;
  char threadCharsetWord[CONST_WORD_LIMIT];
  for(int i = 0; i < CONST_WORD_LIMIT; i++){
    threadCharsetWord[i] = charsetWord[i];
  }
  
  /* Increment current word by thread index */
  next(&threadWordLength, threadCharsetWord, idx);

  char threadTextWord[CONST_WORD_LIMIT];
  for(int i = 0; i < threadWordLength; i++){
    threadTextWord[i] = sharedCharset[threadCharsetWord[i]];
  }
  
  struct md5_context context;
  md5_init(&context, (unsigned char*)threadTextWord, threadWordLength);
  md5Hash(&context);   

  if(context.threadHash[0] == hash01 && context.threadHash[1] == hash02 && context.threadHash[2] == hash03 && context.threadHash[3] == hash04){
    for(int i = 0; i < threadWordLength; i++){
      g_deviceCracked[i] = threadTextWord[i];
    }
    return;
  }
  
}


bool runMD5CUDA(char* words, uint8_t g_wordLength, UINT32* hashBins, bool *result, int *time) {
  // true: found, false: not found
  bool found = false;

  // Start Execution Time
  hipEvent_t start, stop;
  float elapsedTime;
  hipEventCreate(&start); 
  hipEventRecord(start, 0);

  /* Copy current data */
  ERROR_CHECK(hipMemcpy(words, g_word, sizeof(uint8_t) * CONST_WORD_LIMIT, hipMemcpyHostToDevice)); 
  /* Start kernel */
  md5Crack<<<device.max_blocks, device.max_threads>>>(g_wordLength, words, hashBins[0], hashBins[1], hashBins[2], hashBins[3]);
  /* Global increment */
  *result = next(&g_wordLength, g_word, device.max_threads * device.max_blocks);
    
  /* Synchronize now */
  hipDeviceSynchronize();
  /* Copy result */
  ERROR_CHECK(hipMemcpyFromSymbol(g_cracked, HIP_SYMBOL(g_deviceCracked), sizeof(uint8_t) * CONST_WORD_LIMIT, 0, hipMemcpyDeviceToHost)); 
 
  /* Check result */
  if(*g_cracked != 0){     
    std::cout << "Notice: cracked " << g_cracked << std::endl; 
    found = true;
  }

  // Stop Execution Time
  hipEventCreate(&stop);
  hipEventRecord(stop, 0); 
  hipEventSynchronize(stop); 
  hipEventElapsedTime( &elapsedTime, start, stop);
  std::cout << "[DEBUG] elapsedTime = " << elapsedTime << std::endl; 
  *time += elapsedTime;

  return found;
} 

void getOptimalThreads(struct deviceInfo * device) {
	int max_threads = device->prop.maxThreadsPerBlock;
	int max_blocks = 40;
	int shared_memory = device->prop.sharedMemPerBlock - FUNCTION_PARAM_ALLOC;
	
	// calculate the most threads that we can support optimally
	
	while ((shared_memory / max_threads) < REQUIRED_SHARED_MEMORY) { max_threads--; } 	

	device->max_threads = max_threads;		// most threads we support
	device->max_blocks = max_blocks;		// most blocks we support

	// now we need to have (device.max_threads * device.max_blocks) number of words in memory for the graphics card
	
	device->global_memory_len = (device->max_threads * device->max_blocks) * 64;
}


int main(int argc, char* argv[]){

  // FILE *f = fopen("chars.txt", "rb");
  // fseek(f, 0, SEEK_END);
  // long fsize = ftell(f);
  // fseek(f, 0, SEEK_SET);
  
  // char *g_charset = (char *)malloc(fsize + 1);
  // fread(g_charset, fsize, 1, f);
  // fclose(f);
  // g_charset[fsize] = 0;

  int totalTime = 0; 

  device.id = 0;
	hipGetDeviceProperties(&device.prop, device.id);
  getOptimalThreads(&device);

  /* Hash stored as u32 integers */
  UINT32 hashBins[4];
  getHashBins(argv[1], hashBins);
  
  
  /* Fill memory */
  for (int i=0; i<CONST_WORD_LIMIT; i++) {
    g_word[i] = 0;
    g_cracked[i] = 0;
  }
  
  /* Current word length = minimum word length */
  uint8_t g_wordLength = 1;
  
  /* Current word is different on each device */
  char* words;
    
  /* Copy to each device */
  ERROR_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(g_deviceCharset), g_charset, sizeof(uint8_t) * CONST_CHARSET_LENGTH, 0, hipMemcpyHostToDevice));
  ERROR_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(g_deviceCracked), g_cracked, sizeof(uint8_t) * CONST_WORD_LIMIT, 0, hipMemcpyHostToDevice));
  
  /* Allocate on each device */
  ERROR_CHECK(hipMalloc((void**)&words, sizeof(uint8_t) * CONST_WORD_LIMIT));

  bool result = true;
  bool found = false;

  while(result && !found){
    found = runMD5CUDA(words, g_wordLength, hashBins, &result, &totalTime);
  }

  if(!result && !found){
    std::cout << "Notice: found nothing (host)" << std::endl;
  }
    
  /* Free on each device */
  hipFree(words);
  
  std::cout << "Notice: computation time " << totalTime << " ms" << std::endl;
  
}