#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <time.h>
#include <string.h>
#include <stdlib.h>
#include <stdint.h>
#include <sstream>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/device_functions.h>

#include "consts.h"
#include "utility.cu"
#include "md5.cu"
#include "md5.h"

char g_word[CONST_WORD_LIMIT];
char g_charset[] = "abcdefg";
char g_cracked[CONST_WORD_LIMIT];

__device__ char g_deviceCharset[CONST_CHARSET_LENGTH];
__device__ char g_deviceCracked[CONST_WORD_LIMIT];

struct deviceInfo{
	struct hipDeviceProp_t prop; // Device Properties
	int id; // Device ID
	int max_threads; // Device max threads per block
	int max_blocks; // Device max blocks
	int global_memory_len;
};


#define REQUIRED_SHARED_MEMORY 64
#define FUNCTION_PARAM_ALLOC 256
struct deviceInfo device;


__global__ void md5Crack(uint8_t wordLength, char* charsetWord, uint32_t hash01, uint32_t hash02, uint32_t hash03, uint32_t hash04){
  uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  
  /* Shared variables */
  __shared__ char sharedCharset[CONST_CHARSET_LENGTH];
  
  /* Thread variables */
  char threadCharsetWord[CONST_WORD_LIMIT];
  char threadTextWord[CONST_WORD_LIMIT];
  uint8_t threadWordLength = wordLength;
  uint32_t threadHash01, threadHash02, threadHash03, threadHash04;
  
  /* Copy everything to local memory */
  memcpy(threadCharsetWord, charsetWord, CONST_WORD_LIMIT);
  // memcpy(&threadWordLength, &wordLength, sizeof(uint8_t));
  memcpy(sharedCharset, g_deviceCharset, sizeof(uint8_t) * CONST_CHARSET_LENGTH);
  
  /* Increment current word by thread index */
  next(&threadWordLength, threadCharsetWord, idx);
  

  for(uint32_t i = 0; i < threadWordLength; i++){
    threadTextWord[i] = sharedCharset[threadCharsetWord[i]];
  }
  
  struct md5_context context;
  md5_init(&context);

  md5Hash(&context, (unsigned char*)threadTextWord, threadWordLength);   

  if(context.threadHash[0] == hash01 && context.threadHash[1] == hash02 && context.threadHash[2] == hash03 && context.threadHash[3] == hash04){
    for(uint32_t i = 0; i < threadWordLength; i++){
      g_deviceCracked[i] = threadTextWord[i];
    }
    // memcpy(g_deviceCracked, threadTextWord, threadWordLength);
  }
  
}


bool runMD5CUDA(char* words, uint8_t g_wordLength, uint32_t* hashBins, bool *result, int *time) {
  // true: found, false: not found
  bool found = false;

  // Start Execution Time
  hipEvent_t start, stop;
  float elapsedTime;
  hipEventCreate(&start); 
  hipEventRecord(start, 0);

  /* Copy current data */
  ERROR_CHECK(hipMemcpy(words, g_word, sizeof(uint8_t) * CONST_WORD_LIMIT, hipMemcpyHostToDevice)); 
  /* Start kernel */
  md5Crack<<<device.max_blocks, device.max_threads>>>(g_wordLength, words, hashBins[0], hashBins[1], hashBins[2], hashBins[3]);
  /* Global increment */
  *result = next(&g_wordLength, g_word, device.max_threads * device.max_blocks);
    
  /* Synchronize now */
  hipDeviceSynchronize();
  /* Copy result */
  ERROR_CHECK(hipMemcpyFromSymbol(g_cracked, HIP_SYMBOL(g_deviceCracked), sizeof(uint8_t) * CONST_WORD_LIMIT, 0, hipMemcpyDeviceToHost)); 
 
  /* Check result */
  if(*g_cracked != 0){     
    std::cout << "Notice: cracked " << g_cracked << std::endl; 
    found = true;
  }

  // Stop Execution Time
  hipEventCreate(&stop);
  hipEventRecord(stop, 0); 
  hipEventSynchronize(stop); 
  hipEventElapsedTime( &elapsedTime, start, stop);

  *time += elapsedTime;
  return found;
} 

void getOptimalThreads(struct deviceInfo * device) {
	int max_threads;
	int max_blocks;
	int shared_memory;

	max_threads = device->prop.maxThreadsPerBlock;
	shared_memory = device->prop.sharedMemPerBlock - FUNCTION_PARAM_ALLOC;
	
	// calculate the most threads that we can support optimally
	
	while ((shared_memory / max_threads) < REQUIRED_SHARED_MEMORY) { max_threads--; } 

	// now we spread our threads across blocks 
	
	max_blocks = 40;		

	device->max_threads = max_threads;		// most threads we support
	device->max_blocks = max_blocks;		// most blocks we support

	// now we need to have (device.max_threads * device.max_blocks) number of words in memory for the graphics card
	
	device->global_memory_len = (device->max_threads * device->max_blocks) * 64;
}


int main(int argc, char* argv[]){

  // FILE *f = fopen("chars.txt", "rb");
  // fseek(f, 0, SEEK_END);
  // long fsize = ftell(f);
  // fseek(f, 0, SEEK_SET);
  
  // char *g_charset = (char *)malloc(fsize + 1);
  // fread(g_charset, fsize, 1, f);
  // fclose(f);
  // g_charset[fsize] = 0;

  int totalTime = 0; 

  device.id = 0;
	hipGetDeviceProperties(&device.prop, device.id);
  getOptimalThreads(&device);

  /* Hash stored as u32 integers */
  uint32_t hashBins[4];
  getHashBins(argv[1], hashBins);
  
  
  /* Fill memory */
  for (int i=0; i<CONST_WORD_LIMIT; i++) {
    g_word[i] = 0;
    g_cracked[i] = 0;
  }
  
  /* Current word length = minimum word length */
  uint8_t g_wordLength = 1;
  
  /* Current word is different on each device */
  char* words;
    
  /* Copy to each device */
  ERROR_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(g_deviceCharset), g_charset, sizeof(uint8_t) * CONST_CHARSET_LENGTH, 0, hipMemcpyHostToDevice));
  ERROR_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(g_deviceCracked), g_cracked, sizeof(uint8_t) * CONST_WORD_LIMIT, 0, hipMemcpyHostToDevice));
  
  /* Allocate on each device */
  ERROR_CHECK(hipMalloc((void**)&words, sizeof(uint8_t) * CONST_WORD_LIMIT));

  bool result = true;
  bool found = false;

  while(result && !found){
    found = runMD5CUDA(words, g_wordLength, hashBins, &result, &totalTime);
  }

  if(!result && !found){
    std::cout << "Notice: found nothing (host)" << std::endl;
  }
    
  /* Free on each device */
  hipFree(words);
  
  std::cout << "Notice: computation time " << totalTime << " ms" << std::endl;
  
}
