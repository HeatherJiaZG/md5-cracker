#include "hip/hip_runtime.h"
/**
 * CUDA MD5 cracker
 * Copyright (C) 2015  Konrad Kusnierz <iryont@gmail.com>
 *
 * This program is free software; you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation; either version 2 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License along
 * with this program; if not, write to the Free Software Foundation, Inc.,
 * 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.
 */

#include <stdio.h>
#include <iostream>
#include <time.h>
#include <string.h>
#include <stdlib.h>
#include <stdint.h>
#include <sstream>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/device_functions.h>

#include "consts.h"
#include "utility.cu"
#include "md5.cu"

char g_word[CONST_WORD_LIMIT];
char g_charset[CONST_CHARSET_LENGTH];
char g_cracked[CONST_WORD_LIMIT];

__device__ char g_deviceCharset[CONST_CHARSET_LENGTH];
__device__ char g_deviceCracked[CONST_WORD_LIMIT];

__global__ void md5Crack(uint8_t wordLength, char* charsetWord, uint32_t hash01, uint32_t hash02, uint32_t hash03, uint32_t hash04){
  uint32_t idx = (blockIdx.x * blockDim.x + threadIdx.x) * HASHES_PER_KERNEL;
  
  /* Shared variables */
  __shared__ char sharedCharset[CONST_CHARSET_LENGTH];
  
  /* Thread variables */
  char threadCharsetWord[CONST_WORD_LIMIT];
  char threadTextWord[CONST_WORD_LIMIT];
  uint8_t threadWordLength;
  uint32_t threadHash01, threadHash02, threadHash03, threadHash04;
  
  /* Copy everything to local memory */
  memcpy(threadCharsetWord, charsetWord, CONST_WORD_LIMIT);
  memcpy(&threadWordLength, &wordLength, sizeof(uint8_t));
  memcpy(sharedCharset, g_deviceCharset, sizeof(uint8_t) * CONST_CHARSET_LENGTH);
  
  /* Increment current word by thread index */
  next(&threadWordLength, threadCharsetWord, idx);
  
  for(uint32_t hash = 0; hash < HASHES_PER_KERNEL; hash++){
    for(uint32_t i = 0; i < threadWordLength; i++){
      threadTextWord[i] = sharedCharset[threadCharsetWord[i]];
    }
    
    md5Hash((unsigned char*)threadTextWord, threadWordLength, &threadHash01, &threadHash02, &threadHash03, &threadHash04);   

    if(threadHash01 == hash01 && threadHash02 == hash02 && threadHash03 == hash03 && threadHash04 == hash04){
      memcpy(g_deviceCracked, threadTextWord, threadWordLength);
    }
    
    if(!next(&threadWordLength, threadCharsetWord, 1)){
      break;
    }
  }
}


bool runMD5CUDA(char** words, uint8_t g_wordLength, uint32_t* hashBins, bool *result, int *time) {
  // true: found, false: not found
  bool found = false;

  // Start Execution Time
  hipEvent_t start, stop;
  float elapsedTime;
  hipEventCreate(&start); 
  hipEventRecord(start, 0);

  /* Copy current data */
  ERROR_CHECK(hipMemcpy(words[0], g_word, sizeof(uint8_t) * CONST_WORD_LIMIT, hipMemcpyHostToDevice)); 
  /* Start kernel */
  md5Crack<<<TOTAL_BLOCKS, TOTAL_THREADS>>>(g_wordLength, words[0], hashBins[0], hashBins[1], hashBins[2], hashBins[3]);
  /* Global increment */
  *result = next(&g_wordLength, g_word, TOTAL_THREADS * HASHES_PER_KERNEL * TOTAL_BLOCKS);
  
  /* Display progress */
  char word[CONST_WORD_LIMIT];
  
  for(int i = 0; i < g_wordLength; i++){
    word[i] = g_charset[g_word[i]];
  }
    
  /* Synchronize now */
  hipDeviceSynchronize();
  /* Copy result */
  ERROR_CHECK(hipMemcpyFromSymbol(g_cracked, HIP_SYMBOL(g_deviceCracked), sizeof(uint8_t) * CONST_WORD_LIMIT, 0, hipMemcpyDeviceToHost)); 
 
  /* Check result */
  if(*g_cracked != 0){     
    std::cout << "Notice: cracked " << g_cracked << std::endl; 
    found = true;
  }

  // Stop Execution Time
  hipEventCreate(&stop);
  hipEventRecord(stop, 0); 
  hipEventSynchronize(stop); 
  hipEventElapsedTime( &elapsedTime, start, stop);

  *time += elapsedTime;
  return found;
}


int main(int argc, char* argv[]){

  int totalTime = 0; 

  /* Hash stored as u32 integers */
  uint32_t hashBins[4];
  getHashBins(argv[1], hashBins);
  
  /* Fill memory */
  memset(g_word, 0, CONST_WORD_LIMIT);
  memset(g_cracked, 0, CONST_WORD_LIMIT);
  memcpy(g_charset, CONST_CHARSET, CONST_CHARSET_LENGTH);
  
  /* Current word length = minimum word length */
  uint8_t g_wordLength = CONST_WORD_LENGTH_MIN;
  
  
  /* Current word is different on each device */
  char** words = new char*[1];

    
  /* Copy to each device */
  ERROR_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(g_deviceCharset), g_charset, sizeof(uint8_t) * CONST_CHARSET_LENGTH, 0, hipMemcpyHostToDevice));
  ERROR_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(g_deviceCracked), g_cracked, sizeof(uint8_t) * CONST_WORD_LIMIT, 0, hipMemcpyHostToDevice));
  
  /* Allocate on each device */
  ERROR_CHECK(hipMalloc((void**)&words[0], sizeof(uint8_t) * CONST_WORD_LIMIT));

  bool result = true;
  bool found = false;

  while(result && !found){
    found = runMD5CUDA(words, g_wordLength, hashBins, &result, &totalTime);
  }
  
  if(!result && !found){
    std::cout << "Notice: found nothing (host)" << std::endl;
  }
    
  /* Free on each device */
  hipFree((void**)words[0]);
  
  /* Free array */
  delete[] words;
  
  std::cout << "Notice: computation time " << totalTime << " ms" << std::endl;
  
}
